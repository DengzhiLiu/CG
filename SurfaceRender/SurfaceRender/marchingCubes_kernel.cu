#include "hip/hip_runtime.h"
#ifndef _MARCHING_CUBES_KERNEL_CU_
#define _MARCHING_CUBES_KERNEL_CU_

#include <stdio.h>
#include <string.h>
#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include "cutil_math.h"

#include "../SurfaceLib/common.h"
#include "tables.h"

// textures containing look-up tables
texture<uint, 1, hipReadModeElementType> edgeTex;
texture<uint, 1, hipReadModeElementType> triTex;
texture<uint, 1, hipReadModeElementType> numVertsTex;

// volume data
texture<float, 1, hipReadModeElementType> volumeTex;
texture<float, 1, hipReadModeElementType> volumeFvalTex;

extern "C"
void allocateTextures(	uint **d_edgeTable, uint **d_triTable,  uint **d_numVertsTable )
{
    cutilSafeCall(hipMalloc((void**) d_edgeTable, 256*sizeof(uint)));
    cutilSafeCall(hipMemcpy((void *)*d_edgeTable, (void *)edgeTable, 256*sizeof(uint), hipMemcpyHostToDevice) );
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);
    cutilSafeCall(hipBindTexture(0, edgeTex, *d_edgeTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) d_triTable, 256*16*sizeof(uint)));
    cutilSafeCall(hipMemcpy((void *)*d_triTable, (void *)triTable, 256*16*sizeof(uint), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, triTex, *d_triTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) d_numVertsTable, 256*sizeof(uint)));
    cutilSafeCall(hipMemcpy((void *)*d_numVertsTable, (void *)numVertsTable, 256*sizeof(uint), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, numVertsTex, *d_numVertsTable, channelDesc) );
}

extern "C"
void bindVolumeTexture(float *d_volume)
{
    // bind to linear texture
    cutilSafeCall(hipBindTexture(0, volumeTex, d_volume, hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat)));
}

extern "C"
void bindVolumeValTexture(float *d_volumeFval)
{
    // bind to linear texture
    cutilSafeCall(hipBindTexture(0, volumeFvalTex, d_volumeFval, hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat)));
}

// an interesting field function
__device__
float tangle(float x, float y, float z)
{
    x *= 3.0f;
    y *= 3.0f;
    z *= 3.0f;
    return (x*x*x*x - 5.0f*x*x +y*y*y*y - 5.0f*y*y +z*z*z*z - 5.0f*z*z + 11.8f) * 0.2f + 0.5f;
}

// evaluate field function at point
__device__
float fieldFunc(float3 p)
{
    return tangle(p.x, p.y, p.z);
}

// evaluate field function at a point
// returns value and gradient in float4
__device__
float4 fieldFunc4(float3 p)
{
    float v = tangle(p.x, p.y, p.z);
    const float d = 0.001f;
    float dx = tangle(p.x + d, p.y, p.z) - v;
    float dy = tangle(p.x, p.y + d, p.z) - v;
    float dz = tangle(p.x, p.y, p.z + d) - v;
    return make_float4(dx, dy, dz, v);
}

// sample coordinates data set at a pointlist
__device__
float3 sampleVolume(float *data, uint3 p, uint3 gridSize)
{
    p.x = min(p.x, gridSize.x - 1);
    p.y = min(p.y, gridSize.y - 1);
    p.z = min(p.z, gridSize.z - 1);
    uint i = (p.z*gridSize.x*gridSize.y) + (p.y*gridSize.x) + p.x;
//    return (float) data[i] / 255.0f;
    return make_float3(tex1Dfetch(volumeTex, i*3), tex1Dfetch(volumeTex, i*3+1), tex1Dfetch(volumeTex, i*3+2));
}

// sample fval of volume data set at a point
__device__
float sampleVolumeFval(float *data, uint3 p, uint3 gridSize)
{
    p.x = min(p.x, gridSize.x - 1);
    p.y = min(p.y, gridSize.y - 1);
    p.z = min(p.z, gridSize.z - 1);
    uint i = (p.z*gridSize.x*gridSize.y) + (p.y*gridSize.x) + p.x;
//    return (float) data[i] / 255.0f;
    return tex1Dfetch(volumeFvalTex, i);
}

// classify voxel based on number of vertices it will generate
// one thread per voxel
__global__ void
classifyVoxel(uint* voxelVerts, uint* voxelOccupied, float* volumeFval, 
              uint3 gridSize, uint numVoxels, float isoValue)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

	// Data Array A[:, :, 1], A[:, :, 2], ... , A[:, : , n], row first
	uint x = (i % (gridSize.x * gridSize.y)) % gridSize.x;
	uint y = (i % (gridSize.x * gridSize.y)) / gridSize.x;
	uint z = i / (gridSize.x * gridSize.y);
	//uint m = i % (gridSize.x * gridSize.y);
	//uint n = i / (gridSize.x * gridSize.y);
	uint3 gridPos = make_uint3(x, y, z);

    // read field values at neighbouring grid vertices
    float field[8];
    field[0] = sampleVolumeFval(volumeFval, gridPos, gridSize);
    field[1] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 1, 1), gridSize);

    // calculate flag indicating if each vertex is inside or outside isosurface
    uint cubeindex;
	cubeindex =  uint(field[0] < isoValue); 
	cubeindex += uint(field[1] < isoValue)*2; 
	cubeindex += uint(field[2] < isoValue)*4; 
	cubeindex += uint(field[3] < isoValue)*8; 
	cubeindex += uint(field[4] < isoValue)*16; 
	cubeindex += uint(field[5] < isoValue)*32; 
	cubeindex += uint(field[6] < isoValue)*64; 
	cubeindex += uint(field[7] < isoValue)*128;

    // read number of vertices from texture
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);

    if (i < numVoxels) {
        voxelVerts[i] = numVerts;
        voxelOccupied[i] = (numVerts > 0);
    }
}

extern "C" void
launch_classifyVoxel( dim3 grid, dim3 threads, uint* voxelVerts, uint *voxelOccupied, float *volumeFval,
					 uint3 gridSize, uint numVoxels, float isoValue)
{
    // calculate number of vertices need per voxel
    classifyVoxel<<<grid, threads>>>(voxelVerts, voxelOccupied, volumeFval, 
                                     gridSize, numVoxels, isoValue);
    cutilCheckMsg("classifyVoxel failed");
}
              

// compact voxel array
__global__ void
compactVoxels(uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (voxelOccupied[i] && (i < numVoxels)) {
        compactedVoxelArray[ voxelOccupiedScan[i] ] = i;
    }
}

extern "C" void 
launch_compactVoxels(dim3 grid, dim3 threads, uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
{
    compactVoxels<<<grid, threads>>>(compactedVoxelArray, voxelOccupied, 
                                     voxelOccupiedScan, numVoxels);
    cutilCheckMsg("compactVoxels failed");
}

// compute interpolated vertex along an edge
__device__
float3 vertexInterp(float isolevel, float3 p0, float3 p1, float f0, float f1)
{
    float t = (isolevel - f0) / (f1 - f0);
	return lerp(p0, p1, t);
} 

// calculate triangle normal
__device__
float3 calcNormal(float3 *v0, float3 *v1, float3 *v2)
{
    float3 edge0 = *v1 - *v0;
    float3 edge1 = *v2 - *v0;
    // note - it's faster to perform normalization in vertex shader rather than here
    return cross(edge0, edge1);
}

// version that calculates flat surface normal for each triangle
__global__ void
generateTriangles(float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, float *volume,
                  float *volumeFval, uint3 gridSize, float isoValue, uint activeVoxels, uint maxVerts)
{
    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

    if (i > activeVoxels - 1) {
        i = activeVoxels - 1;
    }

#if SKIP_EMPTY_VOXELS
    uint voxel = compactedVoxelArray[i];
#else
    uint voxel = i;
#endif

    // compute position in 3d grid
    uint x = (voxel % (gridSize.x * gridSize.y)) % gridSize.x;
	uint y = (voxel % (gridSize.x * gridSize.y)) / gridSize.x;
	uint z = voxel / (gridSize.x * gridSize.y);
	uint3 gridPos = make_uint3(x, y, z);

	// calculate cell vertex positions
    float3 v[8];
    v[0] = sampleVolume(volume, gridPos, gridSize);
    v[1] = sampleVolume(volume, gridPos + make_uint3(1, 0, 0), gridSize);
    v[2] = sampleVolume(volume, gridPos + make_uint3(1, 1, 0), gridSize);
    v[3] = sampleVolume(volume, gridPos + make_uint3(0, 1, 0), gridSize);
    v[4] = sampleVolume(volume, gridPos + make_uint3(0, 0, 1), gridSize);
    v[5] = sampleVolume(volume, gridPos + make_uint3(1, 0, 1), gridSize);
    v[6] = sampleVolume(volume, gridPos + make_uint3(1, 1, 1), gridSize);
    v[7] = sampleVolume(volume, gridPos + make_uint3(0, 1, 1), gridSize);

    float field[8];
    field[0] = sampleVolumeFval(volumeFval, gridPos, gridSize);
    field[1] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 0, 0), gridSize);
    field[2] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 1, 0), gridSize);
    field[3] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 1, 0), gridSize);
    field[4] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 0, 1), gridSize);
    field[5] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 0, 1), gridSize);
    field[6] = sampleVolumeFval(volumeFval, gridPos + make_uint3(1, 1, 1), gridSize);
    field[7] = sampleVolumeFval(volumeFval, gridPos + make_uint3(0, 1, 1), gridSize);

    // recalculate flag
    uint cubeindex;
	cubeindex =  uint(field[0] < isoValue); 
	cubeindex += uint(field[1] < isoValue)*2; 
	cubeindex += uint(field[2] < isoValue)*4; 
	cubeindex += uint(field[3] < isoValue)*8; 
	cubeindex += uint(field[4] < isoValue)*16; 
	cubeindex += uint(field[5] < isoValue)*32; 
	cubeindex += uint(field[6] < isoValue)*64; 
	cubeindex += uint(field[7] < isoValue)*128;

	// find the vertices where the surface intersects the cube 

#if USE_SHARED
    // use shared memory to avoid using local
	__shared__ float3 vertlist[12*NTHREADS];

	vertlist[threadIdx.x] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[NTHREADS+threadIdx.x] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[(NTHREADS*2)+threadIdx.x] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[(NTHREADS*3)+threadIdx.x] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);
	vertlist[(NTHREADS*4)+threadIdx.x] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[(NTHREADS*5)+threadIdx.x] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[(NTHREADS*6)+threadIdx.x] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[(NTHREADS*7)+threadIdx.x] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);
	vertlist[(NTHREADS*8)+threadIdx.x] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[(NTHREADS*9)+threadIdx.x] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[(NTHREADS*10)+threadIdx.x] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[(NTHREADS*11)+threadIdx.x] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
    __syncthreads();
#else

	float3 vertlist[12];

    vertlist[0] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
    vertlist[1] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
    vertlist[2] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
    vertlist[3] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);

	vertlist[4] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
    vertlist[5] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
    vertlist[6] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
    vertlist[7] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);

	vertlist[8] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
    vertlist[9] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
    vertlist[10] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
    vertlist[11] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
#endif

    // output triangle vertices
    uint numVerts = tex1Dfetch(numVertsTex, cubeindex);
    for(int i=0; i<numVerts; i+=3) {
        uint index = numVertsScanned[voxel] + i;

        float3 *v[3];
        uint edge;
        edge = tex1Dfetch(triTex, (cubeindex*16) + i);
#if USE_SHARED
        v[0] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[0] = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 1);
#if USE_SHARED
        v[1] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[1] = &vertlist[edge];
#endif

        edge = tex1Dfetch(triTex, (cubeindex*16) + i + 2);
#if USE_SHARED
        v[2] = &vertlist[(edge*NTHREADS)+threadIdx.x];
#else
        v[2] = &vertlist[edge];
#endif

        // calculate triangle surface normal
        float3 n = calcNormal(v[0], v[1], v[2]);

        if (index < (maxVerts - 3)) {
            pos[index] = make_float4(*v[0], 1.0f);
            norm[index] = make_float4(n, 0.0f);

            pos[index+1] = make_float4(*v[1], 1.0f);
            norm[index+1] = make_float4(n, 0.0f);

            pos[index+2] = make_float4(*v[2], 1.0f);
            norm[index+2] = make_float4(n, 0.0f);
        }
    }
}

extern "C" void
launch_generateTriangles(dim3 grid, dim3 threads,
						float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned, float *volume,
						float* volumeFval, uint3 gridSize, float isoValue, uint activeVoxels, uint maxVerts)
{
    generateTriangles<<<grid, NTHREADS>>>(pos, norm, 
                                          compactedVoxelArray, 
                                          numVertsScanned, volume, volumeFval,
                                          gridSize, isoValue, activeVoxels, 
                                          maxVerts);
    cutilCheckMsg("generateTriangles failed");
}

#endif
